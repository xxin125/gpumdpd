#include "hip/hip_runtime.h"
#include "fix/fix_wall_reflect.cuh"

/* ----------------------------------------------------------------------------------------------------------- */

wall_reflect::wall_reflect(std::string id, std::string gid, const std::vector<std::string>& params) 
    : Fix(id, gid) {}

/* ----------------------------------------------------------------------------------------------------------- */

std::string wall_reflect::getName() 
{
    return "wall_reflect";
}

/* ----------------------------------------------------------------------------------------------------------- */

void wall_reflect::validateParams(const std::vector<std::string>& params)
{
    std::string error    = "illegal fix_wall_reflect command";
    std::string format   = "fix ID groupID wall_reflect lo/hi wall_position wall_direction";
    std::string example0 = "fix lo_wall water wall_reflect lo 2 z";
    std::string example1 = "fix hi_wall water wall_reflect hi 50 z";
    
    if (params.size() != 3) {
        print_error_and_exit("Invalid fix_wall_reflect parameters", error, format, {example0, example1});
    }

    if (params[0] == "lo") {
        wall_side = 1;
    } else if (params[0] == "hi") {
        wall_side = -1;
    } else {
        print_error_and_exit("Invalid fix_wall_reflect lo/hi", error, format, {example0, example1});
    }

    wall_pos = parse_float<numtyp>(params[1], "Invalid fix_wall_reflect parameters", "position",  {format, example0, example1});

    if (params[2] == "x") {
        wall_direction = 0;
    } else if (params[2] == "y") {
        wall_direction = 1;
    } else if (params[2] == "z") {
        wall_direction = 2;
    } else {
        print_error_and_exit("Invalid fix_wall_reflect direction", error, format, {example0, example1});
    }        
}

/* ----------------------------------------------------------------------------------------------------------- */

void wall_reflect::preprocess(System& system)
{
    Box& box = system.box;
    numtyp box_lo_bound;
    numtyp box_hi_bound;

    if (wall_direction == 0)
    {
        box_lo_bound = box.xlo;
        box_hi_bound = box.xhi;
    }
    else if (wall_direction == 1)
    {
        box_lo_bound = box.ylo;
        box_hi_bound = box.yhi;
    }
    else if (wall_direction == 2)
    {
        box_lo_bound = box.zlo;
        box_hi_bound = box.zhi;
    }

    if (wall_pos < box_lo_bound || wall_pos > box_hi_bound)
    {
        print_error({"Wall position is out of the box!"});
    }
}

/* ----------------------------------------------------------------------------------------------------------- */

static __global__ void kernel_wall_reflect
(
    const int n_gatoms,        const int *g_atoms,     
    numtyp *d_pos,             numtyp *d_uwpos,         numtyp *d_vel,
    const numtyp wall_pos,     const int  wall_side,    const int wall_direction  
)
{
    /* ------------------------------------------------------- */

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n_gatoms) return;
    const int i   = g_atoms[idx];

    /* ------------------------------------------------------- */

    const numtyp gatom_pos = d_pos[i*3+wall_direction];
    const numtyp gatom_vel = d_vel[i*3+wall_direction];
    
    if ((wall_side == 1 && gatom_pos < wall_pos) || (wall_side == -1 && gatom_pos > wall_pos)) {
        numtyp disp = wall_pos - gatom_pos;
        numtyp total_disp = 2 * disp; 
        d_pos[i*3+wall_direction]   += total_disp;
        d_vel[i*3+wall_direction]   = -gatom_vel; 
        d_uwpos[i*3+wall_direction] += total_disp; 
    }
 
    /* ------------------------------------------------------- */
}

/* ----------------------------------------------------------------------------------------------------------- */

void wall_reflect::post_integrate(System& system, unsigned int step) 
{
    /* ------------------------------------------------------- */

    Group& group = find_group(system, this->group_id);
    Atoms& atoms = system.atoms;
    int n_gatoms = group.n_atoms;

    /* ------------------------------------------------------- */

    // reflect 

    int blockSize    = 128;
    int numBlocks    = (n_gatoms + blockSize - 1) / blockSize;

    kernel_wall_reflect<<<numBlocks, blockSize>>>
    (
        n_gatoms,                  group.d_atoms,        
        atoms.d_pos,               atoms.d_uwpos,                 atoms.d_vel,
        wall_pos,                  wall_side,                     wall_direction           
    );
    
    /* ------------------------------------------------------- */
}

/* ----------------------------------------------------------------------------------------------------------- */

///////////////////////////////
REGISTER_FIX(wall_reflect)
///////////////////////////////
